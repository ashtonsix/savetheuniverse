#include "hip/hip_runtime.h"
#include "util.h"
#include "simulation.h"
#include "renderer.h"
#include <hip/hip_runtime.h>
#include <jpeglib.h>
#include <cstdio>

struct Camera
{
    int width;
    int height;
    int quality;
};

unsigned char *frame_buffer = nullptr;
unsigned long frame_size = 0;
unsigned char *d_pixel_data = nullptr;
unsigned char *h_pixel_data = nullptr;

Camera *h_camera = nullptr;
Camera *d_camera = nullptr;

void Renderer::init(int width, int height, int quality)
{
    CHECK_CUDA(hipHostMalloc((void **)&h_pixel_data, width * height * 3 * sizeof(unsigned char)));
    CHECK_CUDA(hipMalloc(&d_pixel_data, width * height * 3 * sizeof(unsigned char)));

    CHECK_CUDA(hipHostMalloc(&h_camera, sizeof(Camera)));
    CHECK_CUDA(hipMalloc(&d_camera, sizeof(Camera)));
    update_camera(width, height, quality);
}

void Renderer::update_camera(int width, int height, int quality)
{
    h_camera->width = width;
    h_camera->height = height;
    h_camera->quality = quality;
    CHECK_CUDA(hipMemcpy(d_camera, h_camera, sizeof(Camera), hipMemcpyHostToDevice));
}

__global__ void k_draw(Camera *camera, float2 *g_state, Simulation::StateConstant *c_state, unsigned char *pixel_data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= camera->width || idy >= camera->height)
        return;

    int pixdex = (idy * c_state->width + idx) * 3;
    float2 value = g_state[idy * c_state->width + idx];

    float mag = hypotf(value.x, value.y) * 1.0f;

    unsigned char color = static_cast<unsigned char>((mag) * 128.0f);
    pixel_data[pixdex] = color;
    pixel_data[pixdex + 1] = color;
    pixel_data[pixdex + 2] = color;
}

void Renderer::draw(unsigned char *&out_buffer, unsigned long &out_size)
{
    using namespace Simulation;

    dim3 dimBlock(16, 16);
    dim3 dimGrid((h_camera->width + dimBlock.x - 1) / dimBlock.x, (h_camera->height + dimBlock.y - 1) / dimBlock.y);
    CHECK_CUDA((k_draw<<<dimGrid, dimBlock>>>(d_camera, d_g_state, d_c_state, d_pixel_data)));
    hipDeviceSynchronize();
    CHECK_CUDA(hipMemcpy(h_pixel_data, d_pixel_data, h_camera->width * h_camera->height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    jpeg_compress_struct cinfo;
    jpeg_error_mgr jerr;
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_compress(&cinfo);

    jpeg_mem_dest(&cinfo, &frame_buffer, &frame_size);

    cinfo.image_width = h_camera->width;
    cinfo.image_height = h_camera->height;
    cinfo.input_components = 3;
    cinfo.in_color_space = JCS_RGB;
    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, h_camera->quality, TRUE);
    jpeg_start_compress(&cinfo, TRUE);

    JSAMPROW row_pointer[1];
    while (cinfo.next_scanline < cinfo.image_height)
    {
        row_pointer[0] = &h_pixel_data[cinfo.next_scanline * h_c_state->width * 3];
        jpeg_write_scanlines(&cinfo, row_pointer, 1);
    }

    jpeg_finish_compress(&cinfo);
    jpeg_destroy_compress(&cinfo);

    out_buffer = frame_buffer;
    out_size = frame_size;
}

void Renderer::cleanup()
{
    hipFree(d_pixel_data);
    hipFree(d_camera);

    hipHostFree(h_pixel_data);
    hipHostFree(h_camera);
    h_pixel_data = nullptr;
    h_camera = nullptr;

    delete[] frame_buffer;
    frame_buffer = nullptr;
    frame_size = 0;
}
