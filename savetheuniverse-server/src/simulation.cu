#include "hip/hip_runtime.h"
#include "util.h"
#include "simulation.h"
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ uint4 random_init(int i, int j)
{
    uint _i = static_cast<uint>(i + 128);
    uint _j = static_cast<uint>(j + 128);
    uint4 random_state = uint4{
        (_i * 1664525 + 1013904223) ^ (_j * 22695477 + 1),
        (_i * 1103515245 + 12345) ^ (_j * 134775813 + 1),
        (_i * 8121 + 28411) ^ (_j * 4096 + 150889),
        (_i * 1229 + 2048) ^ (_j * 279470273 + 0)};

    return random_state;
}

// hybrid taus generator: https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-37-efficient-random-number-generation-and-application
__device__ float random(uint4 &random_state)
{
    random_state.x = ((random_state.x & 4294967294) << 12) ^ (((random_state.x << 13) ^ random_state.x) >> 19);
    random_state.y = ((random_state.y & 4294967288) << 4) ^ (((random_state.y << 2) ^ random_state.y) >> 25);
    random_state.z = ((random_state.z & 4294967280) << 17) ^ (((random_state.z << 3) ^ random_state.z) >> 11);
    random_state.w = 1664525 * random_state.w + 1013904223;

    return 2.3283064365387e-10 * float(random_state.x ^ random_state.y ^ random_state.z ^ random_state.w);
}

float2 *Simulation::d_g_state = nullptr;
Simulation::StateConstant *Simulation::h_c_state = nullptr;
// d_c_state is using global memory. for performance it should be using constant memory, but to make constant memory
// accessible across multiple files (renderer.cu specifically), separable compilation is required (TODO)
Simulation::StateConstant *Simulation::d_c_state = nullptr;

__global__ void k_init(float2 *g_state, Simulation::StateConstant *c_state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    uint4 random_state = random_init(idy * c_state->width + idx, 0);
    float mag = random(random_state);
    float theta = random(random_state) * 2.0f * 3.14159265358979323846f;

    g_state[idy * c_state->width + idx] = float2{mag * cosf(theta), mag * sinf(theta)};
}

void Simulation::init(int width, int height)
{
    CHECK_CUDA(hipMalloc(&d_g_state, width * height * sizeof(float2)));
    CHECK_CUDA(hipMalloc(&d_c_state, sizeof(StateConstant)));
    CHECK_CUDA(hipHostMalloc(&h_c_state, sizeof(StateConstant)));
    update_c_state(width, height);

    dim3 dimBlock(32, 32);
    dim3 dimGrid((h_c_state->width + dimBlock.x - 1) / dimBlock.x, (h_c_state->height + dimBlock.y - 1) / dimBlock.y);
    CHECK_CUDA((k_init<<<dimGrid, dimBlock>>>(d_g_state, d_c_state)));
}

void Simulation::update_c_state(int width, int height)
{

    h_c_state->width = width;
    h_c_state->height = height;

    CHECK_CUDA(hipMemcpy(d_c_state, h_c_state, sizeof(StateConstant), hipMemcpyHostToDevice));
}

__global__ void k_step(float2 *g_state, Simulation::StateConstant *c_state)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int radius = 17;
    int curtain = 16;

    float mag_agg = 0.0f;
    float2 dir_agg = float2{0.0f, 0.0f};

    for (int niy = iy - curtain; niy <= iy + curtain; niy++)
    {
        for (int nix = ix - curtain; nix <= ix + curtain; nix++)
        {
            if (nix == ix && niy == iy)
                continue;

            int niy_torus = (niy + c_state->height) % c_state->height;
            int nix_torus = (nix + c_state->width) % c_state->width;
            float2 rho = g_state[niy_torus * c_state->width + nix_torus];

            // Compute intermediate values
            float2 d = float2{static_cast<float>(nix - ix), static_cast<float>(niy - iy)};
            float dist = hypotf(d.x, d.y);
            float2 d_hat = float2{d.x / dist, d.y / dist};
            float d_dot_rho = d.x * rho.x + d.y * rho.y;
            float rho_magnitude = hypotf(rho.x, rho.y);
            float alpha = d_dot_rho / (dist * rho_magnitude);

            // Compute the distance function X(d)
            float normalized_distance = dist / radius;
            float Xd = max(0.0f, normalized_distance - normalized_distance * normalized_distance);

            // Compute the dispersion function D(alpha)
            float Dalpha = -0.25 * alpha * alpha + 0.5f * alpha + 0.75f;

            // Calculate the pseudo-momentum transfer
            float S = 0.010580575790320963;
            float mag = rho_magnitude * S * Xd * Dalpha;

            // Add to total
            mag_agg += mag;
            dir_agg.x += d_hat.x * mag;
            dir_agg.y += d_hat.y * mag;
        }
    }

    // use aggregates to calculate psuedo-momentum
    float dir_scalar = rhypotf(dir_agg.x, dir_agg.y) * mag_agg;
    float2 rho_prime = float2{dir_agg.x * dir_scalar, dir_agg.y * dir_scalar};

    float2 Q = float2{0.0f, 0.0f};

    // Calculate the constraint vector Q
    float2 center = float2{c_state->width / 2.0f, c_state->height / 2.0f};
    float r = c_state->width * 0.4f;
    float2 d = float2{center.x - ix, center.y - iy};
    float Qi = max(hypotf(d.x, d.y) - r, 0.0f) / 16.0f;
    Q.x = -d.x * Qi;
    Q.y = -d.y * Qi;

    // Apply the constraint vector Q
    float2 rho_prime_hat = float2{rho_prime.x / mag_agg, rho_prime.y / mag_agg};
    float2 modified = float2{Q.x + rho_prime_hat.x, Q.y + rho_prime_hat.y};
    float modified_scalar = rhypotf(modified.x, modified.y);
    modified = float2{modified.x * modified_scalar, modified.y * modified_scalar};
    float2 rho_double_prime = float2{modified.x * mag_agg, modified.y * mag_agg};

    // Update the state
    g_state[iy * c_state->width + ix] = rho_double_prime;
}

void Simulation::step()
{
    dim3 dimBlock(32, 32);
    dim3 dimGrid((h_c_state->width + dimBlock.x - 1) / dimBlock.x, (h_c_state->height + dimBlock.y - 1) / dimBlock.y);
    CHECK_CUDA((k_step<<<dimGrid, dimBlock>>>(d_g_state, d_c_state)));
}

void Simulation::cleanup()
{
    CHECK_CUDA(hipFree(d_g_state));
    CHECK_CUDA(hipFree(d_c_state));

    delete h_c_state;
    h_c_state = nullptr;
}
